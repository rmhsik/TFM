#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <cstdio>
#include <chrono>

typedef std::chrono::high_resolution_clock Clock;

#define ITER 1000000000

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

// CPU version of the vector add function
void vector_add_cpu(int *a, int *b, int *c, int n) {
    int i;

    // Add the vector elements a and b to the vector c
    for (i = 0; i < n; ++i) {
    c[i] = a[i] + b[i];
    }
}

// GPU version of the vector add function
__global__ void vector_add_gpu(int *gpu_a, int *gpu_b, int *gpu_c, int n) {
    int i = blockIdx.x;
    // No for loop needed because the CUDA runtime
    // will thread this ITER times

    gpu_c[i] = gpu_a[i] + gpu_b[i];
}

int main() {

    int *a, *b, *c;
    int *gpu_a, *gpu_b, *gpu_c;
    int *ha, *hb, *hc;

    a = (int *)malloc(ITER * sizeof(int));
    b = (int *)malloc(ITER * sizeof(int));
    c = (int *)malloc(ITER * sizeof(int));
    ha = (int *)malloc(ITER * sizeof(int));
    hb = (int *)malloc(ITER * sizeof(int));
    hc = (int *)malloc(ITER * sizeof(int));
    // We need variables accessible to the GPU,
    // so cudaMallocManaged provides these
    hipMallocManaged(&gpu_a, ITER * sizeof(int));
    hipMallocManaged(&gpu_b, ITER * sizeof(int));
    hipMallocManaged(&gpu_c, ITER * sizeof(int));

    for (int i = 0; i < ITER; ++i) {
        a[i] = i;
        b[i] = i;
        c[i] = i;
    }

    // Call the CPU function and time it
    auto cpu_start = Clock::now();
    vector_add_cpu(a, b, c, ITER);
    auto cpu_end = Clock::now();
    std::cout << "vector_add_cpu: "
    << std::chrono::duration_cast<std::chrono::nanoseconds>(cpu_end - cpu_start).count()
    << " nanoseconds.\n";

    // Call the GPU function and time it
    // The triple angle brakets is a CUDA runtime extension that allows
    // parameters of a CUDA kernel call to be passed.
    // In this example, we are passing one thread block with ITER threads.
    hipMemcpy(gpu_a,a,ITER*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(gpu_b,b,ITER*sizeof(int),hipMemcpyHostToDevice);

    auto gpu_start = Clock::now();
    vector_add_gpu <<<ITER, 1024>>> (gpu_a, gpu_b, gpu_c, ITER);
  
    hipDeviceSynchronize();
    auto gpu_end = Clock::now();
    std::cout << "vector_add_gpu: "
    << std::chrono::duration_cast<std::chrono::nanoseconds>(gpu_end - gpu_start).count()
    << " nanoseconds.\n";
    hipMemcpy(hc,gpu_c,ITER*sizeof(int),hipMemcpyDeviceToHost);
    hipMemcpy(hb,gpu_b,ITER*sizeof(int),hipMemcpyDeviceToHost);
    gpuErrchk(hipMemcpy(ha,gpu_a,ITER*sizeof(int),hipMemcpyDeviceToHost));
    
    std::cout<<c[0]<<"\t"<<hc[0]<<std::endl;
    std::cout<<c[int(ITER/3)]<<"\t"<<hc[int(ITER/3)]<<std::endl;
    std::cout<<c[int(2*ITER/3)]<<"\t"<<hc[int(2*ITER/3)]<<std::endl;
    std::cout<<c[int(ITER-1)]<<"\t"<<hc[int(ITER-1)]<<std::endl;
    // Free the GPU-function based memory allocations
    hipFree(a);
    hipFree(b);
    hipFree(c);

    // Free the CPU-function based memory allocations
    free(a);
    free(b);
    free(c);

    return 0;
}